#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include <Windows.h>
#define N 1000 //N�mero de cuerpos en el universo
#define TIMELAPSE 86400 //N�mero de segundos que pasan entre instantes
#define G 6.67428/100000000000

#define MAXDIM 100000000000 // m
#define MAXSPEED 60000 // m/s
#define MAXMASS 61000000000
#define MINMASS 1000000000
#define randnum(min, max) \
        ((rand() % (int)(((max) + 1) - (min))) + (min))

/*	Orden:
		Nueva Fuerza:
			->Obtener la distancia ortogonal entre A y B
			->Obtener la distancia entre A y B
			->Obtener fuerza diagonal
			->Calcular fuerza ortogonal en base a la fuerza diagonal
			->Obtener Matriz de fuerzas
			->Sumar Matriz de fuerzas
		Nueva Aceleraci�n DONE
		Nueva Posici�n DONE
		Nueva Velocidad DONE
*/



//tama�o cuerpo =
struct cuerpo {
	float pos[2];	//En metros, modificado mediante sumas y restas
	float vel[2];	//En Metros/Segundo, modificado mediante sumas y restas
	float masa;		//En KG, est�tico
	float acel[2];	//En m/s^2, cada iteraci�n es nuevo
	float fuerzas[2]; //En N, cada iteraci�n es nuevo.
};

//Tama�o universo = N*36 = N*Tama�o_Cuerpo
struct universo {
	struct cuerpo cuerpos[N];
};

cuerpo inicializar(cuerpo a, float posicion[2], float velocidad[2], float masa) {
	a.pos[0] = posicion[0];
	a.pos[1] = posicion[1];
	a.vel[0] = velocidad[0];
	a.vel[1] = velocidad[1];
	a.acel[0] = 0;
	a.acel[1] = 0;
	a.fuerzas[0] = 0;
	a.fuerzas[1] = 0;
	a.masa = masa;
	return a;
}

int randomNumber1000(){
	int randomNumber;
	bool checker = true;
	while (checker) {
		randomNumber = rand();
		if (randomNumber <= 1000) {
			checker = false;
		}
	}
	return randomNumber;
}
int randomNumber20000() {
	int randomNumber;
	bool checker = true;
	while (checker) {
		randomNumber = rand();
		if (randomNumber <= 20000) {
			checker = false;
		}
	}
	return randomNumber;
}

float randomPos() {
	float pos1 = 0-MAXDIM;
	float generablePos = (MAXDIM)*2 / 20000;
	float randomNumber = (float)randomNumber20000();
	float pos = pos1 + randomNumber * generablePos;

	return pos;
}

float randomSpeed() {
	float speed1 = 0 - MAXSPEED;
	float generableSpeed = (MAXSPEED) * 2 / 20000;
	float randomNumber = (float)randomNumber20000();
	float speed = speed1 + randomNumber * generableSpeed;

	printf("%f\n", speed);

	return speed;
}

float randomMass() {
	float generableMass = (MAXMASS - MINMASS) / 1000;
	float randomNumber = (float) randomNumber1000();
	float mass = randomNumber * generableMass + MINMASS;
	//printf("GenerableMass: %f\n", generableMass);
	//printf("RandomNumber: %f\n", randomNumber);
	printf("%f\n", mass);
	
	return mass;
}

universo crearUniversoAleatorio(universo* uni) {
	
	cuerpo mundo;
	float vel[2];
	float pos[2];
	float masa;
	for (int i = 0; i < N; i++) {
		masa = randomMass();
		vel[0] = randomSpeed();
		vel[1] = randomSpeed();
		pos[0] = randomPos();
		pos[1] = randomPos();
		uni[0].cuerpos[i] = inicializar(mundo, pos, vel, masa);

	}
	return uni[0];
}

void forceIterate(universo* uni, int idCuerpo1, int idCuerpo2) {

	cuerpo cuerpo1 = uni[0].cuerpos[idCuerpo1];
	cuerpo cuerpo2 = uni[0].cuerpos[idCuerpo2];

	float posX1 = cuerpo1.pos[0];
	float posY1 = cuerpo1.pos[1];
	float posX2 = cuerpo2.pos[0];
	float posY2 = cuerpo2.pos[1];

	float M1 = cuerpo1.masa;
	float M2 = cuerpo2.masa;

	float difX = posX1 - posX2;
	float difY = posY1 - posY2;

	float disTotal = sqrt(difX * difX + difY * difY);

	float F = G * M1 * M2 / (disTotal * disTotal);

	float cos = difX / disTotal;
	float sen = difY / disTotal;

	float Fx = F * cos;
	float Fy = F * sen;

	cuerpo1.fuerzas[0] -= Fx;
	cuerpo1.fuerzas[1] -= Fy;

	cuerpo2.fuerzas[0] += Fx;
	cuerpo2.fuerzas[1] += Fy;

	uni[0].cuerpos[idCuerpo1] = cuerpo1;
	uni[0].cuerpos[idCuerpo2] = cuerpo2;
}

void newForces(universo* uni) {
	//Las fuerzas pasan a ser 0
	for (int i = 0; i < N; i++) {
		uni[0].cuerpos[i].fuerzas[0] = 0;
		uni[0].cuerpos[i].fuerzas[1] = 0;
	}
	for (int i = 0; i < N; i++) {
		for (int j = i + 1; j < N; j++) {
			forceIterate(uni, i, j);
		}
	}
}

void newAcel(universo* uni) {
	float fuerzaX;
	float fuerzaY;
	float masa;
	cuerpo cuerpoActual;
	for (int i = 0; i < N; i++) {
		cuerpoActual = uni[0].cuerpos[i];
		fuerzaX = cuerpoActual.fuerzas[0];
		fuerzaY = cuerpoActual.fuerzas[1];
		masa = cuerpoActual.masa;

		cuerpoActual.acel[0] = fuerzaX / masa;
		cuerpoActual.acel[1] = fuerzaY / masa;
		
		uni[0].cuerpos[i] = cuerpoActual;
	}
}

void newPosition(universo * uni) {
	float velX;
	float velY;
	cuerpo cuerpoActual;

	for (int i = 0; i < N; i++) {
		
		cuerpoActual = uni[0].cuerpos[i];
		velX = cuerpoActual.vel[0];
		velY = cuerpoActual.vel[1];

		cuerpoActual.pos[0] += velX*TIMELAPSE;
		cuerpoActual.pos[1] += velY*TIMELAPSE;

		uni[0].cuerpos[i] = cuerpoActual;
	}
}

void newSpeed (universo* uni) {
	float acelX;
	float acelY;
	cuerpo cuerpoActual;

	for (int i = 0; i < N; i++) {
		cuerpoActual = uni[0].cuerpos[i];
		acelX = cuerpoActual.acel[0];
		acelY = cuerpoActual.acel[1];

		cuerpoActual.vel[0] += acelX * TIMELAPSE;
		cuerpoActual.vel[1] += acelY * TIMELAPSE;

		uni[0].cuerpos[i] = cuerpoActual;
	}
}

void printCuerpos(universo* uni, int iteracion, bool position, bool speed) {
	cuerpo cuerpoActual;
	printf("-------- ITERACION %d --------\n\n", iteracion);
	for (int i = 0; i < N; i++) {
		cuerpoActual = uni[0].cuerpos[i];
		printf("Cuerpo %d:\n\n", i);
		if (position) {
			printf("--Posicion:\n	X:%f\n	Y:%f\n\n", cuerpoActual.pos[0], cuerpoActual.pos[1]);
		}
		if (speed) {
			printf("--Speed:\n	X:%f\n	Y:%f\n\n", cuerpoActual.vel[0], cuerpoActual.vel[1]);
		}

	}
}

void writeData(universo* uni, int iteracion, int nIteracionesTotales) {
	cuerpo cuerpoActual;
	float posX;
	float posY;
	FILE* archivo;
	// Nombre del archivo
	const char* nombreArchivo = "archivo.txt";
	if (iteracion == 0) {
		// Abrir el archivo en modo escritura ("w")
		archivo = fopen(nombreArchivo, "w");
		fprintf(archivo, "%d;%d", nIteracionesTotales, N);
	} else {
		// Abrir el archivo en modo adici�n ("a")
		archivo = fopen(nombreArchivo, "a");
	}

	for (int i = 0; i < N; i++) {
		//Obtener datos
		cuerpoActual = uni[0].cuerpos[i];
		posX = cuerpoActual.pos[0];
		posY = cuerpoActual.pos[1];

		fprintf(archivo, "\n%d;%d;%f;%f", iteracion, i, posX, posY); 
		//fprintf(archivo, "\n%f;%f", posX, posY);
		//Imprimir en formato X;Y
	}

	fclose(archivo);


}

void iterateUniverse(universo* uni, int nSegundos, bool print) {
	int timeLeft = nSegundos;
	int nIteration = 0;
	int nIteracionesTotales = nSegundos / TIMELAPSE;
	while (timeLeft >= TIMELAPSE) {
		if (print) {
			printCuerpos(uni, nIteration, true, true);
			writeData(uni, nIteration, nIteracionesTotales+1);
		}
		newForces(uni);
		newAcel(uni);
		newPosition(uni);
		newSpeed(uni);
		timeLeft -= TIMELAPSE; 
		nIteration++;
	}
	if (print) {
		printCuerpos(uni, nIteration, true, true);
		writeData(uni, nIteration, nIteracionesTotales+1);
	}
}

int main() {

	//printf("Tama�o cuerpo: %d\n", sizeof(universo));

	struct universo* uni = (universo*)malloc(sizeof(universo));
	uni[0] = crearUniversoAleatorio(uni);
	iterateUniverse(uni, 31536000, true);

	return 0;
}
