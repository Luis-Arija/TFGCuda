#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
#include <Windows.h>
#define N 5 //N�mero de cuerpos en el universo
#define TIMELAPSE 1 //N�mero de segundos que pasan entre instantes
#define NELEMS(x)  (sizeof(x) / sizeof((x)[0]))
#define G 6.67428/100000000000

/*	Orden:
		Nueva Fuerza:
			->Obtener la distancia ortogonal entre A y B
			->Obtener la distancia entre A y B
			->Obtener fuerza diagonal
			->Calcular fuerza ortogonal en base a la fuerza diagonal
			->Obtener Matriz de fuerzas
			->Sumar Matriz de fuerzas
		Nueva Aceleraci�n DONE
		Nueva Posici�n DONE
		Nueva Velocidad DONE
*/



//tama�o cuerpo =
struct cuerpo {
	float pos[2];	//En metros, modificado mediante sumas y restas
	float vel[2];	//En Metros/Segundo, modificado mediante sumas y restas
	float masa;		//En KG, est�tico
	float acel[2];	//En m/s^2, cada iteraci�n es nuevo
	float fuerzas[2]; //En N, cada iteraci�n es nuevo.
};

//Tama�o universo = N*36 = N*Tama�o_Cuerpo
struct universo {
	struct cuerpo cuerpos[N];
};

cuerpo inicializar(cuerpo a, float posicion[2], float velocidad[2], float masa) {
	a.pos[0] = posicion[0];
	a.pos[1] = posicion[1];
	a.vel[0] = velocidad[0];
	a.vel[1] = velocidad[1];
	a.acel[0] = 0;
	a.acel[1] = 0;
	a.fuerzas[0] = 0;
	a.fuerzas[1] = 0;
	a.masa = masa;
	return a;
}

void forceIterate(universo* uni, int idCuerpo1, int idCuerpo2) {

	cuerpo cuerpo1 = uni[0].cuerpos[idCuerpo1];
	cuerpo cuerpo2 = uni[0].cuerpos[idCuerpo2];

	float posX1 = cuerpo1.pos[0];
	float posY1 = cuerpo1.pos[1];
	float posX2 = cuerpo2.pos[0];
	float posY2 = cuerpo2.pos[1];

	float M1 = cuerpo1.masa;
	float M2 = cuerpo2.masa;

	float difX = posX1 - posX2;
	float difY = posY1 - posY2;

	float disTotal = sqrt(difX * difX + difY * difY);

	float F = G * M1 * M2 / (disTotal * disTotal);

	float cos = difX / disTotal;
	float sen = difY / disTotal;

	float Fx = F * cos;
	float Fy = F * sen;

	cuerpo1.fuerzas[0] -= Fx;
	cuerpo1.fuerzas[1] -= Fy;

	cuerpo2.fuerzas[0] += Fx;
	cuerpo2.fuerzas[1] += Fy;

	uni[0].cuerpos[idCuerpo1] = cuerpo1;
	uni[0].cuerpos[idCuerpo2] = cuerpo2;
}

void newForces(universo* uni) {
	//Las fuerzas pasan a ser 0
	for (int i = 0; i < N; i++) {
		uni[0].cuerpos[i].fuerzas[0] = 0;
		uni[0].cuerpos[i].fuerzas[1] = 0;
	}
	for (int i = 0; i < N; i++) {
		for (int j = i + 1; j < N; j++) {
			forceIterate(uni, i, j);
		}
	}
}

void newAcel(universo* uni) {
	float fuerzaX;
	float fuerzaY;
	float masa;
	cuerpo cuerpoActual;
	for (int i = 0; i < N; i++) {
		cuerpoActual = uni[0].cuerpos[i];
		fuerzaX = cuerpoActual.fuerzas[0];
		fuerzaY = cuerpoActual.fuerzas[1];
		masa = cuerpoActual.masa;

		cuerpoActual.acel[0] = fuerzaX / masa;
		cuerpoActual.acel[1] = fuerzaY / masa;
		
		uni[0].cuerpos[i] = cuerpoActual;
	}
}

void newPosition(universo * uni) {
	float velX;
	float velY;
	cuerpo cuerpoActual;

	for (int i = 0; i < N; i++) {
		
		cuerpoActual = uni[0].cuerpos[i];
		velX = cuerpoActual.vel[0];
		velY = cuerpoActual.vel[1];

		cuerpoActual.pos[0] += velX*TIMELAPSE;
		cuerpoActual.pos[1] += velY*TIMELAPSE;

		uni[0].cuerpos[i] = cuerpoActual;
	}
}

void newSpeed (universo* uni) {
	float acelX;
	float acelY;
	cuerpo cuerpoActual;

	for (int i = 0; i < N; i++) {
		cuerpoActual = uni[0].cuerpos[i];
		acelX = cuerpoActual.acel[0];
		acelY = cuerpoActual.acel[1];

		cuerpoActual.vel[0] += acelX * TIMELAPSE;
		cuerpoActual.vel[1] += acelY * TIMELAPSE;

		uni[0].cuerpos[i] = cuerpoActual;
	}
}

void printCuerpos(universo* uni, int iteracion, bool position, bool speed) {
	cuerpo cuerpoActual;
	printf("-------- ITERACION %d --------\n\n", iteracion);
	for (int i = 0; i < N; i++) {
		cuerpoActual = uni[0].cuerpos[i];
		printf("Cuerpo %d:\n\n", i);
		if (position) {
			printf("--Posicion:\n	X:%f\n	Y:%f\n\n", cuerpoActual.pos[0], cuerpoActual.pos[1]);
		}
		if (speed) {
			printf("--Speed:\n	X:%f\n	Y:%f\n\n", cuerpoActual.vel[0], cuerpoActual.vel[1]);
		}

	}
}

void writeData(universo* uni, int iteracion, int nIteracionesTotales) {
	cuerpo cuerpoActual;
	float posX;
	float posY;
	FILE* archivo;
	// Nombre del archivo
	const char* nombreArchivo = "archivo.txt";
	if (iteracion == 0) {
		// Abrir el archivo en modo escritura ("w")
		archivo = fopen(nombreArchivo, "w");
		fprintf(archivo, "%d;%d", nIteracionesTotales, N);
	} else {
		// Abrir el archivo en modo adici�n ("a")
		archivo = fopen(nombreArchivo, "a");
	}

	for (int i = 0; i < N; i++) {
		//Obtener datos
		cuerpoActual = uni[0].cuerpos[i];
		posX = cuerpoActual.pos[0];
		posY = cuerpoActual.pos[1];

		fprintf(archivo, "\n%d;%d;%f;%f", iteracion, i, posX, posY); 
		//fprintf(archivo, "\n%f;%f", posX, posY);
		//Imprimir en formato X;Y
	}

	fclose(archivo);


}

void iterateUniverse(universo* uni, int nSegundos, bool print) {
	int timeLeft = nSegundos;
	int nIteration = 0;
	int nIteracionesTotales = nSegundos / TIMELAPSE;
	while (timeLeft >= TIMELAPSE) {
		if (print) {
			printCuerpos(uni, nIteration, true, true);
			writeData(uni, nIteration, nIteracionesTotales+1);
		}
		newForces(uni);
		newAcel(uni);
		newPosition(uni);
		newSpeed(uni);
		timeLeft -= TIMELAPSE; 
		nIteration++;
	}
	if (print) {
		printCuerpos(uni, nIteration, true, true);
		writeData(uni, nIteration, nIteracionesTotales+1);
	}
}

int main() {

	//printf("Tama�o cuerpo: %d\n", sizeof(universo));

	struct cuerpo mundo;
	struct universo* uni = (universo*)malloc(sizeof(universo));
	
	float posicion[] = { 0,0 };
	float posicion2[] = { 10,10 };
	float posicion3[] = { 10,-10 };
	float posicion4[] = { -10,-10 };
	float posicion5[] = { -10,10 };
	float velocidad[] = { 0, 0 };
	float masa = 1000000000000;

	uni[0].cuerpos[0] = inicializar(mundo, posicion, velocidad, masa);
	uni[0].cuerpos[1] = inicializar(mundo, posicion2, velocidad, masa);
	uni[0].cuerpos[2] = inicializar(mundo, posicion3, velocidad, masa);
	uni[0].cuerpos[3] = inicializar(mundo, posicion4, velocidad, masa);
	uni[0].cuerpos[4] = inicializar(mundo, posicion5, velocidad, masa);
	iterateUniverse(uni, 200, true);

	return 0;
}
