#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include <Windows.h>
#define N 10//N�mero de cuerpos en el universo. MAXIMO 14000
#define nNiveles 1 //N�mero de niveles. M�ximo 9 por tama�o int
//tendr�n una dim de MAXDIM/pow(2,nNiveles)

#define TIMELAPSE 86400 //N�mero de segundos que pasan entre instantes
#define G 6.67428/100000000000
#define NELEMS(x)  (sizeof(x) / sizeof((x)[0]))
#define MAXDIM 15*pow(10, 8)
#define MAXSPEED 30000 // m/s
#define MAXMASS 6*pow(10,24)
#define MINMASS 1*pow(10,23)
#define CLEANTREEITERATION 5
#define randnum(min, max) \
        ((rand() % (int)(((max) + 1) - (min))) + (min))

/*	Orden:
		Nueva Fuerza:
			->Obtener la distancia ortogonal entre A y B
			->Obtener la distancia entre A y B
			->Obtener fuerza diagonal
			->Calcular fuerza ortogonal en base a la fuerza diagonal
			->Obtener Matriz de fuerzas
			->Sumar Matriz de fuerzas
		Nueva Aceleraci�n DONE
		Nueva Posici�n DONE
		Nueva Velocidad DONE
*/



//tama�o cuerpo =
struct cuerpo {
	float pos[2];	//En metros, modificado mediante sumas y restas
	float vel[2];	//En Metros/Segundo, modificado mediante sumas y restas
	float masa;		//En KG, est�tico
	float acel[2];	//En m/s^2, cada iteraci�n es nuevo
	float fuerzas[2]; //En N, cada iteraci�n es nuevo.
};

//Tama�o universo = N*36 = N*Tama�o_Cuerpo
struct universo {
	struct cuerpo cuerpos[N];
};

cuerpo inicializar(cuerpo a, float posicion[2], float velocidad[2], float masa) {
	a.pos[0] = posicion[0];
	a.pos[1] = posicion[1];
	a.vel[0] = velocidad[0];
	a.vel[1] = velocidad[1];
	a.acel[0] = 0;
	a.acel[1] = 0;
	a.fuerzas[0] = 0;
	a.fuerzas[1] = 0;
	a.masa = masa;
	return a;
}

int randomNumber1000(){
	int randomNumber;
	bool checker = true;
	while (checker) {
		randomNumber = rand();
		if (randomNumber <= 1000) {
			checker = false;
		}
	}
	return randomNumber;
}
int randomNumber20000() {
	int randomNumber;
	bool checker = true;
	while (checker) {
		randomNumber = rand();
		if (randomNumber <= 20000) {
			checker = false;
		}
	}
	return randomNumber;
}

float randomPos() {
	float pos1 = 0-MAXDIM;
	float generablePos = (MAXDIM)*2 / 20000;
	float randomNumber = (float)randomNumber20000();
	float pos = pos1 + randomNumber * generablePos;

	return pos;
}

float randomSpeed() {
	float speed1 = 0 - MAXSPEED;
	float generableSpeed = (MAXSPEED) * 2 / 20000;
	float randomNumber = (float)randomNumber20000();
	float speed = speed1 + randomNumber * generableSpeed;

	//printf("%f\n", speed);

	return speed;
}

float randomMass() {
	float generableMass = (MAXMASS - MINMASS) / 1000;
	float randomNumber = (float) randomNumber1000();
	float mass = randomNumber * generableMass + MINMASS;
	//printf("GenerableMass: %f\n", generableMass);
	//printf("RandomNumber: %f\n", randomNumber);
	//printf("%f\n", mass);
	
	return mass;
}

//Le pasas un puntero malloc y lo rellena
void crearUniversoAleatorio(universo* uni) {

	struct cuerpo a;
	for (int i = 0; i < N; i++) {
		a = uni->cuerpos[i];
		a.masa = randomMass();
		a.vel[0] = randomSpeed();
		a.vel[1] = randomSpeed();
		a.pos[0] = randomPos();
		a.pos[1] = randomPos();
		a.acel[0] = 0;
		a.acel[1] = 0;
		a.fuerzas[0] = 0;
		a.fuerzas[1] = 0;
		uni->cuerpos[i] = a;

	}

	//return uni[0];
}

void forceIterate(universo* uni, int idCuerpo1, int idCuerpo2) {

	cuerpo cuerpo1 = uni[0].cuerpos[idCuerpo1];
	cuerpo cuerpo2 = uni[0].cuerpos[idCuerpo2];

	float posX1 = cuerpo1.pos[0];
	float posY1 = cuerpo1.pos[1];
	float posX2 = cuerpo2.pos[0];
	float posY2 = cuerpo2.pos[1];

	float M1 = cuerpo1.masa;
	float M2 = cuerpo2.masa;

	float difX = posX1 - posX2;
	float difY = posY1 - posY2;

	float disTotal = sqrt(difX * difX + difY * difY);

	float F = G * M1 * M2 / (disTotal * disTotal);

	float cos = difX / disTotal;
	float sen = difY / disTotal;

	float Fx = F * cos;
	float Fy = F * sen;

	cuerpo1.fuerzas[0] -= Fx;
	cuerpo1.fuerzas[1] -= Fy;

	cuerpo2.fuerzas[0] += Fx;
	cuerpo2.fuerzas[1] += Fy;

	uni[0].cuerpos[idCuerpo1] = cuerpo1;
	uni[0].cuerpos[idCuerpo2] = cuerpo2;
}

void newForces(universo* uni) {
	//Las fuerzas pasan a ser 0
	for (int i = 0; i < N; i++) {
		uni[0].cuerpos[i].fuerzas[0] = 0;
		uni[0].cuerpos[i].fuerzas[1] = 0;
	}
	for (int i = 0; i < N; i++) {
		for (int j = i + 1; j < N; j++) {
			forceIterate(uni, i, j);
		}
	}
}

void newAcel(universo* uni) {
	float fuerzaX;
	float fuerzaY;
	float masa;
	float acelX;
	float acelY;
	cuerpo cuerpoActual;
	for (int i = 0; i < N; i++) {
		cuerpoActual = uni[0].cuerpos[i];
		fuerzaX = cuerpoActual.fuerzas[0];
		fuerzaY = cuerpoActual.fuerzas[1];
		masa = cuerpoActual.masa;
		acelX = fuerzaX / masa;
		acelY = fuerzaY / masa;
		cuerpoActual.acel[0] = acelX;
		cuerpoActual.acel[1] = acelY;
		
		uni[0].cuerpos[i] = cuerpoActual;
	}
}

void newPosition(universo * uni) {
	float velX;
	float velY;
	cuerpo cuerpoActual;

	for (int i = 0; i < N; i++) {
		
		cuerpoActual = uni[0].cuerpos[i];
		velX = cuerpoActual.vel[0];
		velY = cuerpoActual.vel[1];

		cuerpoActual.pos[0] += velX*TIMELAPSE;
		cuerpoActual.pos[1] += velY*TIMELAPSE;

		uni[0].cuerpos[i] = cuerpoActual;
	}
}

void newSpeed (universo* uni) {
	float acelX;
	float acelY;
	cuerpo cuerpoActual;

	for (int i = 0; i < N; i++) {
		cuerpoActual = uni[0].cuerpos[i];
		acelX = cuerpoActual.acel[0];
		acelY = cuerpoActual.acel[1];

		cuerpoActual.vel[0] += acelX * TIMELAPSE;
		cuerpoActual.vel[1] += acelY * TIMELAPSE;

		uni[0].cuerpos[i] = cuerpoActual;
	}
}

void printCuerpos(universo* uni, int iteracion, bool position, bool speed) {
	cuerpo cuerpoActual;
	printf("-------- ITERACION %d --------\n\n", iteracion);
	for (int i = 0; i < N; i++) {
		cuerpoActual = uni[0].cuerpos[i];
		printf("Cuerpo %d:\n\n", i);
		if (position) {
			printf("--Posicion:\n	X:%f\n	Y:%f\n\n", cuerpoActual.pos[0], cuerpoActual.pos[1]);
		}
		if (speed) {
			printf("--Speed:\n	X:%f\n	Y:%f\n\n", cuerpoActual.vel[0], cuerpoActual.vel[1]);
		}

	}
}

void writeData(universo* uni, int iteracion, int nIteracionesTotales) {
	cuerpo cuerpoActual;
	float posX;
	float posY;
	FILE* archivo;
	// Nombre del archivo
	const char* nombreArchivo = "archivo.txt";
	if (iteracion == 0) {
		// Abrir el archivo en modo escritura ("w")
		archivo = fopen(nombreArchivo, "w");
		fprintf(archivo, "%d;%d", nIteracionesTotales, N);
	} else {
		// Abrir el archivo en modo adici�n ("a")
		archivo = fopen(nombreArchivo, "a");
	}

	for (int i = 0; i < N; i++) {
		//Obtener datos
		cuerpoActual = uni[0].cuerpos[i];
		posX = cuerpoActual.pos[0];
		posY = cuerpoActual.pos[1];

		fprintf(archivo, "\n%d;%d;%f;%f", iteracion, i, posX, posY); 
		//fprintf(archivo, "\n%f;%f", posX, posY);
		//Imprimir en formato X;Y
	}

	fclose(archivo);


}

void iterateUniverse(universo* uni, int nSegundos, bool print) {
	int timeLeft = nSegundos;
	int nIteration = 0;
	int nIteracionesTotales = nSegundos / TIMELAPSE;
	while (timeLeft >= TIMELAPSE) {
		if (print) {
			printCuerpos(uni, nIteration, true, true);
			writeData(uni, nIteration, nIteracionesTotales+1);
		}
		newForces(uni);
		newAcel(uni);
		newPosition(uni);
		newSpeed(uni);
		timeLeft -= TIMELAPSE; 
		nIteration++;
		printf("llego a antes de newAccel\n");
		printf("Iteracion n: %d\n", nIteration);
	}
	if (print) {
		printCuerpos(uni, nIteration, true, true);
		writeData(uni, nIteration, nIteracionesTotales+1);
	}
}

int main() {

	//printf("Tama�o cuerpo: %d\n", sizeof(universo));

	struct universo* uni = (universo*)malloc(sizeof(universo));
	uni = new universo;
	crearUniversoAleatorio(uni);
	printCuerpos(uni, 0, true, true);
	iterateUniverse(uni, 864000, false);
	//printCuerpos(uni, 11, true, true);

	return 0;
}
